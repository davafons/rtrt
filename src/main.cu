#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <thread>

#include <SDL.h>
#include <hiprand/hiprand_kernel.h>

#include "frontend/texturegpu.cuh"
#include "frontend/window.hpp"
#include "hitable/hitable_list.cuh"
#include "hitable/sphere.cuh"
#include "kernels/kernels.cuh"
#include "math/camera.cuh"
#include "math/ray.cuh"
#include "math/vec3.cuh"
#include "utils/config.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/managed_ptr.cuh"

template <typename... Args>
void launch_2D_texture_kernel(void (*kernel)(TextureGPU *, Args...),
                              const Config &config, TextureGPU *tex,
                              Args... args) {
  dim3 blocks = config.blocks(tex->get_width(), tex->get_height());

  kernel<<<blocks, config.threads>>>(tex, std::forward<Args>(args)...);
  cudaCheckErr(hipGetLastError());

  tex->copy_to_cpu();
}

__global__ void create_world(HitableList **hitable_objects) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if ((x == 0) && (y == 0)) {
    float R = cos(M_PI / 4);
    *hitable_objects = new HitableList();
    (*hitable_objects)->push_back(new Sphere(Vec3(0, 0, -1), 0.5f));
    (*hitable_objects)->push_back(new Sphere(Vec3(0, -100.5f, -1), 100));
    (*hitable_objects)->push_back(new Sphere(Vec3(1, 0, -1), 0.5f));
    (*hitable_objects)->push_back(new Sphere(Vec3(-1, 0, -1), 0.5f));
    (*hitable_objects)->push_back(new Sphere(Vec3(-1, 0, -1), -0.45f));
  }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if ((x >= max_x) || (y >= max_y)) {
    return;
  }

  int pixel_index = y * max_x + x;

  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

template <class T, typename... Args>
__global__ void push_back(HitableList **hitable_objects, Args... args) {
  (*hitable_objects)->push_back(new T(args...));
}

void input_thread_task(Window &window, Camera &camera) {
  while (!window.should_quit()) {
    SDL_Event e;
    while (SDL_PollEvent(&e) != 0) {
      if (e.type == SDL_QUIT) {
        window.close();
      }
    }

    const Uint8 *state = SDL_GetKeyboardState(NULL);

    if (state[SDL_SCANCODE_W]) {
      camera.move(Camera::Movement::FORWARD, window.get_delta_time());
    }

    if (state[SDL_SCANCODE_S]) {
      camera.move(Camera::Movement::BACKWARD, window.get_delta_time());
    }

    if (state[SDL_SCANCODE_A]) {
      camera.move(Camera::Movement::LEFT, window.get_delta_time());
    }

    if (state[SDL_SCANCODE_D]) {
      camera.move(Camera::Movement::RIGHT, window.get_delta_time());
    }

    if (state[SDL_SCANCODE_Q]) {
      camera.move(Camera::Movement::DOWN, window.get_delta_time());
    }

    if (state[SDL_SCANCODE_E]) {
      camera.move(Camera::Movement::UP, window.get_delta_time());
    }
  }
}

int main() {
  Config gConfig;

  {
    Window window("Raytracer", 800, 400);
    Camera gCamera(Vec3(-2, 2, 1), Vec3(0, 0, -1), Vec3(0, 1, 0), 90,
                   float(window.get_width()) / window.get_height());

    managed_ptr<TextureGPU> viewport = make_managed<TextureGPU>(
        window.get_renderer(), window.get_width(), window.get_height(), 0.75f);

    HitableList **hitable_objects =
        cuda_malloc<HitableList *>(sizeof(HitableList *));
    create_world<<<1, 1>>>(hitable_objects);

    cudaCheckErr(hipDeviceSynchronize());
    cudaCheckErr(hipGetLastError());

    hiprandState *d_rand_state = cuda_malloc<hiprandState>(
        viewport->get_n_pixels() * sizeof(hiprandState));
    dim3 blocks = gConfig.blocks(viewport->get_width(), viewport->get_height());

    render_init<<<blocks, gConfig.threads>>>(
        viewport->get_width(), viewport->get_height(), d_rand_state);

    cudaCheckErr(hipDeviceSynchronize());
    cudaCheckErr(hipGetLastError());

    /* push_back<Sphere><<<1, 1>>>(hitable_objects, Vec3(-1, 0, -1), 0.5f); */
    /* cudaCheckErr(hipDeviceSynchronize()); */
    /* cudaCheckErr(hipGetLastError()); */

    std::thread input_thread(input_thread_task, std::ref(window),
                             std::ref(gCamera));

    gCamera.set_ns(20);

    int frames = 0;
    float time = 0.0f;
    float avg_fps = 0.0f;

    while (!window.should_quit()) {
      window.update_delta_time();

      window.clear_render();

      launch_2D_texture_kernel(chapter_7_kernel, gConfig, viewport.get(),
                               gCamera, (Hitable **)hitable_objects,
                               d_rand_state);

      viewport->copy_to_renderer(window.get_renderer());

      window.present_render();

      time += window.get_delta_time();
      ++frames;
      avg_fps += window.get_fps();

      if (time >= 0.5f) {
        std::cout << avg_fps / frames << std::endl;

        time = 0.0f;
        frames = 0;
        avg_fps = 0;
      }
    }

    input_thread.join();
  }

  cudaCheckErr(hipDeviceReset());

  return 0;
}
