#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <thread>

#include <SDL.h>
#include <hiprand/hiprand_kernel.h>

#include "frontend/texturegpu.cuh"
#include "frontend/window.hpp"
#include "hitable/hitable_list.cuh"
#include "hitable/sphere.cuh"
#include "kernels/kernels.cuh"
#include "math/camera.cuh"
#include "math/ray.cuh"
#include "math/vec3.cuh"
#include "utils/config.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/managed_ptr.cuh"

template <typename... Args>
void launch_2D_texture_kernel(void (*kernel)(TextureGPU *, Args...),
                              const Config &config, TextureGPU *tex,
                              Args... args) {
  dim3 blocks = config.blocks(tex->get_width(), tex->get_height());

  kernel<<<blocks, config.threads>>>(tex, std::forward<Args>(args)...);
  cudaCheckErr(hipGetLastError());

  tex->copy_to_cpu();
}

__global__ void create_world(HitableList **hitable_objects) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if ((x == 0) && (y == 0)) {
    float R = cos(M_PI / 4);
    *hitable_objects = new HitableList();
    (*hitable_objects)->push_back(new Sphere(Vec3(0, 0, -1), 0.5f));
    (*hitable_objects)->push_back(new Sphere(Vec3(0, -100.5f, -1), 100));
    (*hitable_objects)->push_back(new Sphere(Vec3(1, 0, -1), 0.5f));
    (*hitable_objects)->push_back(new Sphere(Vec3(-1, 0, -1), 0.5f));
    (*hitable_objects)->push_back(new Sphere(Vec3(-1, 0, -1), -0.45f));
  }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if ((x >= max_x) || (y >= max_y)) {
    return;
  }

  int pixel_index = y * max_x + x;

  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

template <class T, typename... Args>
__global__ void push_back(HitableList **hitable_objects, Args... args) {
  (*hitable_objects)->push_back(new T(args...));
}

int main() {
  Config gConfig;

  {
    Window window("Raytracer", 800, 400);
    Camera gCamera(Vec3(-2, 2, 1), Vec3(0, 0, -1), Vec3(0, 1, 0), 90,
                   float(window.get_width()) / window.get_height());

    managed_ptr<TextureGPU> viewport = make_managed<TextureGPU>(
        window.get_renderer(), window.get_width(), window.get_height(), 0.75f);

    HitableList **hitable_objects =
        cuda_malloc<HitableList *>(sizeof(HitableList *));
    create_world<<<1, 1>>>(hitable_objects);

    cudaCheckErr(hipDeviceSynchronize());
    cudaCheckErr(hipGetLastError());

    hiprandState *d_rand_state = cuda_malloc<hiprandState>(
        viewport->get_n_pixels() * sizeof(hiprandState));
    dim3 blocks = gConfig.blocks(viewport->get_width(), viewport->get_height());

    render_init<<<blocks, gConfig.threads>>>(
        viewport->get_width(), viewport->get_height(), d_rand_state);

    cudaCheckErr(hipDeviceSynchronize());
    cudaCheckErr(hipGetLastError());

    /* push_back<Sphere><<<1, 1>>>(hitable_objects, Vec3(-1, 0, -1), 0.5f); */
    /* cudaCheckErr(hipDeviceSynchronize()); */
    /* cudaCheckErr(hipGetLastError()); */

    int ns = 20;
    int MAX_NS = 200;
    gCamera.set_ns(ns);

    while (!window.should_quit()) {
      SDL_Event e;
      while (SDL_PollEvent(&e) != 0) {
        if (e.type == SDL_QUIT) {
          window.close();
        }

        switch (e.key.keysym.sym) {

        case SDLK_ESCAPE:
          window.close();
          break;

        case SDLK_w:
          gCamera.move(Camera::Movement::FORWARD, window.get_delta_time());
          break;

        case SDLK_s:
          gCamera.move(Camera::Movement::BACKWARD, window.get_delta_time());
          break;

        case SDLK_d:
          gCamera.move(Camera::Movement::RIGHT, window.get_delta_time());
          break;

        case SDLK_a:
          gCamera.move(Camera::Movement::LEFT, window.get_delta_time());
          break;

        case SDLK_e:
          gCamera.move(Camera::Movement::UP, window.get_delta_time());
          break;

        case SDLK_q:
          gCamera.move(Camera::Movement::DOWN, window.get_delta_time());
          break;
        }
      }

      window.update_delta_time();

      window.clear_render();

      if (ns < MAX_NS) {
        launch_2D_texture_kernel(chapter_7_kernel, gConfig, viewport.get(),
                                 gCamera, (Hitable **)hitable_objects,
                                 d_rand_state);
        /* ns += 20; */
        /* gCamera.set_ns(ns); */
      }

      /* gCamera.move_right(0.01f): */

      viewport->copy_to_renderer(window.get_renderer());

      window.present_render();

      std::cout << window.get_fps() << std::endl;
    }
  }

  cudaCheckErr(hipDeviceReset());

  return 0;
}
