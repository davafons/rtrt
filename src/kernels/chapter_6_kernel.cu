#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <cfloat>

#include "frontend/texturegpu.cuh"
#include "hitable/hitable_list.cuh"
#include "math/camera.cuh"
#include "math/ray.cuh"
#include "math/vec3.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/managed_ptr.cuh"
#include "utils/world.cuh"

__device__ float hit_sphere_6(const Vec3 &center, float radius, const Ray &r) {
  Vec3 oc = r.origin() - center;
  float a = dot(r.direction(), r.direction());
  float b = 2.0f * dot(oc, r.direction());
  float c = dot(oc, oc) - radius * radius;

  float discriminant = b * b - 4 * a * c;
  if (discriminant < 0) {
    return -1.0f;
  } else {
    return (-b - sqrt(discriminant)) / (2.0f * a);
  }
}

__device__ Vec3 color_6(const Ray &r, Hitable **hitable_objects) {
  HitRecord rec;

  if ((*hitable_objects)->hit(r, 0.0f, 10.0f, rec)) {

    return 0.5f *
           Vec3(rec.normal.x() + 1, rec.normal.y() + 1, rec.normal.z() + 1);

  } else {
    Vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0f);

    return (1.0f - t) * Vec3(1.0f, 1.0f, 1.0f) + t * Vec3(0.5f, 0.7f, 1.0f);
  }
}

__global__ void chapter_6_kernel(TextureGPU *tex, Camera camera,
                                 Hitable **hitable_objects,
                                 hiprandState *rand_state) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  size_t w = tex->get_width();
  size_t h = tex->get_height();

  if ((x >= w || (y >= h)))
    return;

  int pixel_index = y * w + x;
  hiprandState local_rand_state = rand_state[pixel_index];

  Vec3 col(0.0f, 0.0f, 0.0f);

  for (int s = 0; s < camera.get_ns(); ++s) {
    float u = float(x + hiprand_uniform(&local_rand_state)) / float(w);
    float v = float(h - y + hiprand_uniform(&local_rand_state)) / float(h);

    Ray ray = camera.get_ray(u, v);

    col += color_6(ray, hitable_objects);
  }

  col /= float(camera.get_ns());

  Uint8 r = col.r() * 255.99f;
  Uint8 g = col.g() * 255.99f;
  Uint8 b = col.b() * 255.99f;

  tex->set_rgb(x, y, r, g, b);
}
