#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include <cfloat>

#include "frontend/texturegpu.cuh"
#include "hitable/hitable_list.cuh"
#include "math/camera.cuh"
#include "math/math.cuh"
#include "math/ray.cuh"
#include "math/vec3.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/managed_ptr.cuh"
#include "utils/world.cuh"

__device__ Vec3 color(const Ray &r, Hitable **hitable_objects,
                      hiprandState *local_rand_state) {
  Ray cur_ray = r;
  Vec3 cur_attenuation = Vec3(1.0f, 1.0f, 1.0f);

  for (int i = 0; i < 50; ++i) {
    HitRecord rec;
    if ((*hitable_objects)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
      Ray scattered;
      Vec3 attenuation;

      if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered,
                               local_rand_state)) {
        cur_attenuation *= attenuation;
        cur_ray = scattered;
      } else {
        return Vec3(0.0f, 0.0f, 0.0f);
      }

    } else {
      Vec3 unit_direction = unit_vector(cur_ray.direction());
      float t = 0.5f * (unit_direction.y() + 1.0f);

      Vec3 c = (1.0f - t) * Vec3(1.0f, 1.0f, 1.0f) + t * Vec3(0.5f, 0.7f, 1.0f);

      return cur_attenuation * c;
    }
  }

  return Vec3(0.0f, 0.0f, 0.0f);
}

__global__ void raytracing(TextureGPU *tex, Camera camera,
                           Hitable **hitable_objects, hiprandState *rand_state) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  size_t w = tex->get_width();
  size_t h = tex->get_height();

  if ((x >= w || (y >= h)))
    return;

  int pixel_index = y * w + x;
  hiprandState *local_rand_state = &rand_state[pixel_index];

  Vec3 col(0.0f, 0.0f, 0.0f);

  for (int s = 0; s < camera.get_ns(); ++s) {
    float u = float(x + hiprand_uniform(local_rand_state)) / float(w);
    float v = float(h - y + hiprand_uniform(local_rand_state)) / float(h);

    Ray ray = camera.get_ray(u, v, local_rand_state);

    col += color(ray, hitable_objects, local_rand_state);
  }

  rand_state[pixel_index] = *local_rand_state;

  col /= float(camera.get_ns());
  Uint8 r = sqrt(col.r()) * 255.99f;
  Uint8 g = sqrt(col.g()) * 255.99f;
  Uint8 b = sqrt(col.b()) * 255.99f;

  tex->set_rgb(x, y, r, g, b);
}
