#include "viewport.cuh"

Viewport::Viewport(SDL_Renderer *renderer, size_t w, size_t h,
                   Uint32 pixel_format_enum)
    : renderer_(renderer), width_(w), height_(h) {
  tex_ = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
                           SDL_TEXTUREACCESS_STREAMING, w, h);

  hipMallocManaged((void **)&fmt_, sizeof(SDL_PixelFormat));

  SDL_PixelFormat *tmp_pixelformat = SDL_AllocFormat(pixel_format_enum);
  hipMemcpy(fmt_, tmp_pixelformat, sizeof(SDL_PixelFormat),
             hipMemcpyHostToDevice);

  hipMalloc((void **)&d_pixels_, width_ * height_ * sizeof(Uint32));
}

void Viewport::lock() {
  SDL_LockTexture(tex_, NULL, &pixels_, &pitch_);
  locked_ = true;
}

void Viewport::lock_gpu() {
  lock();

  hipMemcpy(d_pixels_, pixels_, width_ * height_ * sizeof(Uint32),
             hipMemcpyHostToDevice);
}

void Viewport::unlock() {
  SDL_UnlockTexture(tex_);
  pixels_ = NULL;
  pitch_ = 0;
  locked_ = false;
}

void Viewport::unlock_gpu() {
  hipMemcpy(pixels_, d_pixels_, width_ * height_ * sizeof(Uint32),
             hipMemcpyDeviceToHost);

  unlock();
}

Uint32 &Viewport::access(int x, int y) {
  return static_cast<Uint32 *>(d_pixels_)[y * width_ + x];
}

void Viewport::set_rgb(int x, int y, Uint8 r, Uint8 g, Uint8 b) {
  access(x, y) = (r >> fmt_->Rloss) << fmt_->Rshift |
                 (g >> fmt_->Gloss) << fmt_->Gshift |
                 (b >> fmt_->Bloss) << fmt_->Bshift | fmt_->Amask;
}

void Viewport::set_rgba(int x, int y, Uint8 r, Uint8 g, Uint8 b, Uint8 a) {
  access(x, y) = 0xFFFFFFFF;
  /* access(x, y) = (r >> fmt_->Rloss) << fmt_->Rshift | */
  /*                (g >> fmt_->Gloss) << fmt_->Gshift | */
  /*                (b >> fmt_->Bloss) << fmt_->Bshift | */
  /*                ((a >> fmt_->Aloss) << fmt_->Ashift & fmt_->Amask); */
}
